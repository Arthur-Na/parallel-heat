#include "heat_simulator_cuda.hh"

#include <iomanip>

int main(int argc, char** argv)
{
  if (argc != 4) {
    std::cerr << "Usage : ./heat-diffusion [input_file] [output_file] [nb_iter]" << std::endl;
    return 1;
  }

  long nb_iter = std::stoi(argv[3]);
  if (nb_iter < 0) {
    std::cerr << "Usage : ./heat-diffusion [input_file] [output_file] [nb_iter]" << std::endl
      << "nb_iter must be positive." << std::endl;
    return 1;
  }

  std::streamsize ss = std::cout.precision();

  auto simulator = HeatSimulator(argv[1]);


  hipEvent_t start;
  hipEvent_t stop;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  float *res = simulator.simulate_cuda(nb_iter);

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));

  float elapsedTime;
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
  printf( "Time to generate: %3.1f ms\n", elapsedTime );
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  //std::cout << std::setprecision(2);
//  for (unsigned i = 0; i < simulator.get_size(); ++i)
//    std::cout << std::fixed << std::setprecision(2) << res[i] << " ";
//  std::cout << std::endl;

  return 0;
}

#include "hip/hip_runtime.h"
#include "heat_simulator_cuda.hh"

HeatSimulator::HeatSimulator(std::string input_file)
{
  std::ifstream input(input_file);

  long x, y, z;

  float default_initial_value = 0;

  input >> x_ >> y_ >> z_ >> default_initial_value;

  //mesh_ = std::vector<float>(x_ * y_ * z_, default_initial_value);
  mesh_ = new float[x_ * y_ * z_];
  for (unsigned i = 0; i < x_ * y_ * z_; ++i)
    mesh_[i] = default_initial_value;

  float next_value;
  while(input >> x >> y >> z >> next_value)
  {
    //mesh_.at((x * x_ +  y) * y_ + z) = next_value;
    mesh_[(x * x_ +  y) * y_ + z] = next_value;
  }

  //max_value_ = *std::max_element(mesh_.begin(), mesh_.end());
}

HeatSimulator::~HeatSimulator()
{
  delete mesh_;
}

__device__ float kernel_computeDX(float *mesh, int idx, int idy, int idz, int sx, int sy)
{
  float a = idx + 1 < sx ? mesh[((idx + 1) * sx + idy) * sy + idz] : 0;
  float b = 2 * mesh[(idx * sx + idy) * sy + idz];
  float c = idx - 1 >= 0 ? mesh[((idx - 1) * sx + idy) * sy + idz] : 0;
  return a - b + c;
}

__device__ float kernel_computeDY(float *mesh, int idx, int idy, int idz, int sx, int sy)
{
  float a = idy + 1 < sy ? mesh[(idx * sx + (idy + 1)) * sy + idz] : 0;
  float b = 2 * mesh[(idx * sx + idy) * sy + idz];
  float c = idy - 1 >= 0 ? mesh[(idx * sx + (idy - 1)) * sy + idz] : 0;
  return a - b + c;
}

__device__ float kernel_computeDZ(float *mesh, int idx, int idy, int idz, int sx, int sy, int sz)
{
  float a = idz + 1 < sz ? mesh[(idx * sx + idy) * sy + idz + 1] : 0;
  float b = 2 * mesh[(idx * sx + idy) * sy + idz];
  float c = idz - 1 >= 0 ? mesh[(idx * sx + idy) * sy + idz - 1] : 0;
  return a - b + c;
}

__device__ float kernel_computeD(float *mesh, int idx, int idy, int idz, int sx, int sy, int sz)
{
  if (idx != 0 && idx != sx - 1 && idy != 0 && idy != sy - 1 && idz != 0 && idz != sz - 1)
    return kernel_computeDX(mesh, idx, idy, idz, sx, sy)
           + kernel_computeDY(mesh, idx, idy, idz, sx, sy)
           + kernel_computeDZ(mesh, idx, idy, idz, sx, sy, sz);
  else
    return 0;
}

__global__ void kernel_compute(float* mesh_in, int sx, int sy, int sz, float* mesh_out)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int idz = blockIdx.z * blockDim.z + threadIdx.z;

  if (idx < sx && idy < sy && idz < sz) 
  {
    const float alpha = 0.000019;
    int index = (idx * sx + idy) * sy + idz; 
    
    //mesh_out[idx][idy][idz] = mesh_in[idx][idy][idz] + alpha * kernel_computeD(mesh_in, idx, idy, idz);
    mesh_out[index] = mesh_in[index] + alpha * kernel_computeD(mesh_in, idx, idy, idz, sx, sy, sz);
  }
}


#define BLKXSIZE 32
#define BLKYSIZE 8
#define BLKZSIZE 4

//__constant__ float* iter_n;

//void HeatSimulator::simulate_cuda(unsigned max_iter, float* mesh_ref, float *mesh_out)
float* HeatSimulator::simulate_cuda(unsigned max_iter)
{
  int SIZE = x_ * y_ * z_;
  const dim3 block_size(BLKXSIZE, BLKYSIZE, BLKZSIZE);
  const dim3 grid_size(((x_ + BLKXSIZE - 1) / BLKXSIZE),
                       ((y_ + BLKYSIZE - 1) / BLKYSIZE),
                       ((z_ + BLKZSIZE - 1) / BLKZSIZE));


  float* new_mesh = mesh_;
  float* mesh_out;
  HANDLE_ERROR(hipMalloc((void**) &mesh_out, SIZE * sizeof(float)));
  for (unsigned n = 1; n < max_iter; ++n)
  {
    //__constant__ float iter_n[SIZE];
    float* iter_n;
    HANDLE_ERROR(hipMalloc((void**) &iter_n, SIZE * sizeof(float)));
    HANDLE_ERROR(hipMemcpy(iter_n, new_mesh, SIZE * sizeof(float), hipMemcpyHostToDevice));
    //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(iter_n), new_mesh, SIZE * sizeof(float)));

    kernel_compute<<<grid_size, block_size>>>(iter_n, x_, y_, z_, mesh_out);
    
    HANDLE_ERROR(hipMemcpy(new_mesh, mesh_out, SIZE * sizeof(float), hipMemcpyDeviceToHost));
  }

  HANDLE_ERROR(hipFree(mesh_out));
  return new_mesh;
}
